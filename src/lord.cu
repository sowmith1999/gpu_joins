#include "lord.cuh"
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

void copyGHashRelContainerToDevice(GHashRelContainer* &d_container, const GHashRelContainer &h_container) {
    hipMalloc(&d_container, sizeof(GHashRelContainer));

    MEntity* d_index_map = nullptr;
    hipMalloc(&d_index_map, sizeof(MEntity) * h_container.index_map_size);
    hipMemcpy(d_index_map, h_container.index_map, sizeof(MEntity) * h_container.index_map_size, hipMemcpyHostToDevice);

    GHashRelContainer temp_container = h_container;
    temp_container.index_map = d_index_map;

    hipMemcpy(d_container, &temp_container, sizeof(GHashRelContainer), hipMemcpyHostToDevice);
}

int main() {
    const int numTables = 2; // Example number of tables
    std::vector<GHashRelContainer*> h_tables(numTables);
    std::vector<GHashRelContainer*> d_tables(numTables); // Pointers to device tables

    for(int i = 0; i < numTables; ++i) {
        hipMalloc(&d_tables[i], sizeof(GHashRelContainer));
        hipMalloc(&d_tables[i]->index_map, sizeof(MEntity) * h_tables[i]->index_map_size);
        hipMemcpy(d_tables[i]->index_map, h_tables[i]->index_map, sizeof(MEntity) * h_tables[i]->index_map_size, hipMemcpyHostToDevice);
    }

    GHashRelContainer** d_tablesArray;
    hipMalloc(&d_tablesArray, numTables * sizeof(GHashRelContainer*));
    hipMemcpy(d_tablesArray, d_tables.data(), numTables * sizeof(GHashRelContainer*), hipMemcpyHostToDevice);

    unsigned int* d_matchingCounts;
    hipMalloc(&d_matchingCounts, sizeof(unsigned int));
    hipMemset(d_matchingCounts, 0, sizeof(unsigned int));

    const int threadsPerBlock = 256;
    const int blocksPerGrid = (h_tables[0]->index_map_size + threadsPerBlock - 1) / threadsPerBlock;
    for (int i = 0; i < numTables; ++i) {
        GHashRelContainer* d_table = d_tables[i];
        k_aryJoinKernel<<<blocksPerGrid, threadsPerBlock>>>(d_table, 1, reinterpret_cast<tuple_size_t*>(d_matchingCounts));
    }


    unsigned int h_matchingCounts;
    hipMemcpy(&h_matchingCounts, d_matchingCounts, sizeof(unsigned int), hipMemcpyDeviceToHost);

    std::cout << "Total matching tuples across all tables: " << h_matchingCounts << std::endl;

    hipFree(d_matchingCounts);
    for(auto t : d_tables) {
        hipFree(t);
    }
    hipFree(d_tablesArray);

    return 0;
}

// void executeGHashRelJoinKernel(GHashRelContainer &table1, GHashRelContainer &table2) {
//     GHashRelContainer *d_table1, *d_table2;
//     int *d_output, *d_matchCount;
//     int matchCount = 0;
//     int *output = new int[table1.index_map_size]; // Maximum possible matches

//     hipMalloc(&d_output, table1.index_map_size * sizeof(int));
//     hipMalloc(&d_matchCount, sizeof(int));
//     hipMemset(d_matchCount, 0, sizeof(int));

//     copyGHashRelContainerToDevice(d_table1, table1);
//     copyGHashRelContainerToDevice(d_table2, table2);

//     int threadsPerBlock = 256;
//     int blocksPerGrid = (table1.index_map_size + threadsPerBlock - 1) / threadsPerBlock;

//     GHashRelJoinKernel<<<blocksPerGrid, threadsPerBlock>>>(d_table1, d_table2, d_output, d_matchCount);

//     hipMemcpy(output, d_output, table1.index_map_size * sizeof(int), hipMemcpyDeviceToHost);
//     hipMemcpy(&matchCount, d_matchCount, sizeof(int), hipMemcpyDeviceToHost);

//     std::cout << "Match count: " << matchCount << std::endl;
//     for(int i = 0; i < matchCount; ++i) {
//         std::cout << "Match found at index " << output[i] << " in Table 1\n";
//     }

//     hipFree(d_table1->index_map);
//     hipFree(d_table2->index_map);
//     hipFree(d_table1);
//     hipFree(d_table2);
//     hipFree(d_output);
//     hipFree(d_matchCount);
//     delete[] output;
// }

// int main() {
//     const size_t table1Size = 3, table2Size = 3;
//     MEntity table1Data[table1Size] = {{1, 0}, {2, 0}, {3, 0}};
//     MEntity table2Data[table2Size] = {{2, 0}, {3, 0}, {4, 0}};

//     GHashRelContainer table1(0, 0, 0), table2(0, 0, 0);
//     table1.index_map = table1Data;
//     table1.index_map_size = table1Size;
//     table2.index_map = table2Data;
//     table2.index_map_size = table2Size;

//     executeGHashRelJoinKernel(table1, table2);

//     return 0;
// }