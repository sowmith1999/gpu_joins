#include "graph_utilities.cuh"
#include "gpu_join_kernels.cuh"
#include <iostream>
#include <vector>

int main() {
    // These are graphs where we have source nodes and dest nodes. So source(1) and dest(2) = edge(1, 2)
    // in this current setup, there is edge(1, 2) in the first index. 
    std::vector<std::vector<int>> srcNodesVec = {
        {1, 2, 4}, 
        {1, 11, 2}, 
        {1, 10, 3}  
    };

    std::vector<std::vector<int>> destNodesVec = {
        {2, 3, 2}, 
        {2, 3, 2}, 
        {2, 3, 4}  
    };

    int numEdges = srcNodesVec[0].size();

    Graph* graphsOnGPU;
    hipMallocManaged(&graphsOnGPU, 3 * sizeof(Graph)); 

    for (int i = 0; i < 3; ++i) {
        new (&graphsOnGPU[i]) Graph(numEdges); 
        fillGraphData(graphsOnGPU[i], srcNodesVec[i], destNodesVec[i]);
    }

    int* outputBuffer;
    hipMalloc(&outputBuffer, numEdges * sizeof(int));

    int blockSize = 256; 
    int numBlocks = (numEdges + blockSize - 1) / blockSize;

    intersectGraphsKernel<<<numBlocks, blockSize>>>(graphsOnGPU, 3, outputBuffer, numEdges);
    hipDeviceSynchronize();

    std::vector<int> intersectionResults(numEdges);
    hipMemcpy(intersectionResults.data(), outputBuffer, numEdges * sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Intersection Results:" << std::endl;
    for (int i = 0; i < numEdges; ++i) {
        std::cout << intersectionResults[i] << (i < numEdges - 1 ? ", " : "\n");
    }

    // Cleanup
    for (int i = 0; i < 3; ++i) {
        hipFree(graphsOnGPU[i].srcNodes);
        hipFree(graphsOnGPU[i].destNodes);
    }
    hipFree(graphsOnGPU);
    hipFree(outputBuffer);

    return 0;
}