// Authors: Michael G & Sowmith K
#include "kernels.cuh"
#include <iostream>
#include <vector>

int main() {
    // These are graphs where we have source nodes and dest nodes. So source(1) and dest(2) = edge(1, 2)
    // in this current setup, there is edge(1, 2) in the first index. 
    std::vector<std::vector<int>> srcNodesVec = {
        {1, 2, 4}, 
        {1, 11, 2}, 
        {1, 10, 3}  
    };

    std::vector<std::vector<int>> destNodesVec = {
        {2, 3, 2}, 
        {2, 3, 2}, 
        {2, 3, 4}  
    };

    int numEdges = srcNodesVec[0].size();

    Graph* graphsOnGPU;
    hipMallocManaged(&graphsOnGPU, 3 * sizeof(Graph)); 

    for (int i = 0; i < 3; ++i) {
        new (&graphsOnGPU[i]) Graph(numEdges); 
        fillGraphData(graphsOnGPU[i], srcNodesVec[i], destNodesVec[i]);
    }

    int* outputBuffer;
    hipMalloc(&outputBuffer, numEdges * sizeof(int));

    int blockSize = 1;
    int numBlocks = (numEdges + blockSize - 1) / blockSize;

    int* intersectionCount;
    hipMalloc(&intersectionCount, sizeof(int));
    hipMemset(intersectionCount, 0, sizeof(int));

    intersectGraphsKernel<<<numBlocks, blockSize>>>(graphsOnGPU, 3, outputBuffer, numEdges, intersectionCount);
    hipDeviceSynchronize();

    int h_intersectionCount;
    hipMemcpy(&h_intersectionCount, intersectionCount, sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Total Intersections: " << h_intersectionCount << std::endl;
    std::vector<int> intersectionResults(numEdges);
    hipMemcpy(intersectionResults.data(), outputBuffer, numEdges * sizeof(int), hipMemcpyDeviceToHost);

    // Merging area
    Graph* outputGraph;
    hipMallocManaged(&outputGraph, sizeof(Graph));
    hipMallocManaged(&outputGraph->srcNodes, h_intersectionCount * sizeof(int));
    hipMallocManaged(&outputGraph->destNodes, h_intersectionCount * sizeof(int));
    outputGraph->numEdges = h_intersectionCount;

    numBlocks = (outputGraph->numEdges + blockSize - 1) / blockSize;

    mergeGraphsKernel<<<numBlocks, blockSize>>>(graphsOnGPU, 3, outputGraph);
    hipDeviceSynchronize();

    std::cout << "Output Graph: " << std::endl;
    for (int i = 0; i < outputGraph->numEdges; ++i) {
        // Ensure you are checking against a valid sentinel value or ensure edges are initialized correctly
        if (outputGraph->srcNodes[i] != -1 && outputGraph->destNodes[i] != -1) {
            std::cout << "(" << outputGraph->srcNodes[i] << ", " << outputGraph->destNodes[i] << ")" << std::endl;
        }
    }

//    int intersectionCount = /* result from counting intersections */;
//    hipMallocManaged(&outputGraph->srcNodes, intersectionCount * sizeof(int));
//    hipMallocManaged(&outputGraph->destNodes, intersectionCount * sizeof(int));

    // std::cout << "Intersection Results:" << std::endl;
    // for (int i = 0; i < numEdges; ++i) {
    //     std::cout << intersectionResults[i] << (i < numEdges - 1 ? ", " : "\n");
    // }

    // Cleanup
    hipFree(graphsOnGPU->srcNodes);
    hipFree(graphsOnGPU->destNodes);
    hipFree(outputGraph->srcNodes);
    hipFree(outputGraph->destNodes);
    hipFree(outputGraph);
    hipFree(graphsOnGPU);
    hipFree(outputBuffer);
    hipFree(intersectionCount);

    return 0;
}