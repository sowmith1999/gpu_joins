// Authors: Michael G & Sowmith K
#include "kernels.cuh"
#include <iostream>
#include <vector>

int main() {
    // These are graphs where we have source nodes and dest nodes. So source(1) and dest(2) = edge(1, 2)
    // in this current setup, there is edge(1, 2) in the first index. 
    std::vector<std::vector<int>> srcNodesVec = {
        {1, 2, 4}, 
        {1, 11, 2}, 
        {1, 10, 3}  
    };

    std::vector<std::vector<int>> destNodesVec = {
        {2, 3, 2}, 
        {2, 3, 2}, 
        {2, 3, 4}  
    };

    int numEdges = srcNodesVec[0].size();

    Graph* graphsOnGPU;
    hipMallocManaged(&graphsOnGPU, 3 * sizeof(Graph)); 

    for (int i = 0; i < 3; ++i) {
        new (&graphsOnGPU[i]) Graph(numEdges); 
        fillGraphData(graphsOnGPU[i], srcNodesVec[i], destNodesVec[i]);
    }

    int* outputBuffer;
    hipMalloc(&outputBuffer, numEdges * sizeof(int));

    int blockSize = 256; 
    int numBlocks = (numEdges + blockSize - 1) / blockSize;

    intersectGraphsKernel<<<numBlocks, blockSize>>>(graphsOnGPU, 3, outputBuffer, numEdges);
    hipDeviceSynchronize();

    Graph* outputGraphOnGPU;
    hipMallocManaged(&outputGraphOnGPU, numEdges * sizeof(Graph));

    mergeGraphsKernel<<<numBlocks, blockSize>>>(graphsOnGPU, 3, outputGraphOnGPU);
    hipDeviceSynchronize();

    // print output graph
    for (int i = 0; i < numEdges; ++i) {
        std::cout << "Edge " << i << ": ";
        for (int j = 0; j < outputGraphOnGPU[i].numEdges; ++j) {
            std::cout << outputGraphOnGPU[i].srcNodes[j] << " -> " << outputGraphOnGPU[i].destNodes[j] << (j < outputGraphOnGPU[i].numEdges - 1 ? ", " : "\n");
        }
    }

    // std::vector<int> intersectionResults(numEdges);
    // hipMemcpy(intersectionResults.data(), outputBuffer, numEdges * sizeof(int), hipMemcpyDeviceToHost);

    // std::cout << "Intersection Results:" << std::endl;
    // for (int i = 0; i < numEdges; ++i) {
    //     std::cout << intersectionResults[i] << (i < numEdges - 1 ? ", " : "\n");
    // }

    // Cleanup
    for (int i = 0; i < 3; ++i) {
        hipFree(graphsOnGPU[i].srcNodes);
        hipFree(graphsOnGPU[i].destNodes);
    }
    hipFree(graphsOnGPU);
    hipFree(outputBuffer);

    return 0;
}