// Authors: Michael G & Sowmith K
#include <iostream>
#include <chrono>
// #include <vector> // should come from graph.cuh
#include "kernels.cuh"
#include "graph.cuh"
// #include "graph.h.gch"

int main() {
    // These are graphs where we have source nodes and dest nodes. So source(1) and dest(2) = edge(1, 2)
    // in this current setup, there is edge(1, 2) in the first index.
    // std::vector<std::vector<int>> srcNodesVec = {
    //         {1, 2, 4},
    //         {1, 11, 2},
    //         {1, 10, 3}
    // };

    // std::vector<std::vector<int>> destNodesVec = {
    //         {2, 3, 2},
    //         {2, 3, 2},
    //         {2, 3, 4}
    // };

    int numEdges = srcNodesVec[0].size();

    Graph* graphsOnGPU;
    hipMallocManaged(&graphsOnGPU, 3 * sizeof(Graph));

    for (int i = 0; i < 3; ++i) {
        new (&graphsOnGPU[i]) Graph(numEdges);
        fillGraphData(graphsOnGPU[i], srcNodesVec[i], destNodesVec[i]);
    }

    Graph* outputGraph;
    hipMallocManaged(&outputGraph, sizeof(Graph));
    hipMallocManaged(&outputGraph->srcNodes, numEdges * sizeof(int));
    hipMallocManaged(&outputGraph->destNodes, numEdges * sizeof(int));
    outputGraph->numEdges = numEdges;

    int blockSize = 1;
    int numBlocks = (numEdges + blockSize - 1) / blockSize;

    int* offsetter;
    hipMalloc(&offsetter, sizeof(int));
    hipMemset(offsetter, 0, sizeof(int));

    auto combinedStart = std::chrono::steady_clock::now();

    intersectAndMergeGraphsKernel<<<numBlocks, blockSize>>>(graphsOnGPU, 3, outputGraph, numEdges, offsetter);
    hipDeviceSynchronize();

    auto combinedEnd = std::chrono::steady_clock::now();

    std::cout << "Output Graph: " << std::endl;
    for (int i = 0; i < outputGraph->numEdges; ++i) {
        if (outputGraph->srcNodes[i] != -1 && outputGraph->destNodes[i] != -1) {
            std::cout << "(" << outputGraph->srcNodes[i] << ", " << outputGraph->destNodes[i] << ")" << std::endl;
        }
    }

    std::cout << "Time (nanoseconds): " << std::chrono::duration<double, std::milli>(combinedEnd - combinedStart).count() << " ms" << std::endl;

    // Cleanup
    hipFree(graphsOnGPU->srcNodes);
    hipFree(graphsOnGPU->destNodes);
    hipFree(outputGraph->srcNodes);
    hipFree(outputGraph->destNodes);
    hipFree(outputGraph);
    hipFree(graphsOnGPU);
    hipFree(offsetter);

    return 0;
}