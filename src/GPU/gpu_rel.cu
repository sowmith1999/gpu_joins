#include "hip/hip_runtime.h"
#include "../../gdlog/include/tuple.cuh"
#include <algorithm>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <random>
#include <sstream>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <thrust/unique.h>
#include <vector>

#define DEBUG 0

/********************************/
const uint32_t kHashTableCapacity = 524288;
const uint32_t kEmpty = 0xffffffff;

__device__ uint32_t hash(uint32_t key) {
  key ^= key >> 16;
  key *= 0x85ebca6b;
  key ^= key >> 13;
  key *= 0xc2b2ae35;
  key ^= key >> 16;
  return key & (kHashTableCapacity - 1);
}

typedef struct KeyValue {
  uint32_t key;
  uint32_t value;
} KeyValue;

void checkCUDAError(const char* msg) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

__device__ void insertKey(KeyValue* hashtable, uint32_t key, uint32_t value) {
  uint32_t slot = hash(key);
  while (true) {
    uint32_t prev = atomicCAS(&hashtable[slot].key, kEmpty, key);
    if (prev == kEmpty || prev == key) {
      hashtable[slot].value = value;
      return;
    }
    slot = (slot + 1) & (kHashTableCapacity - 1);
  }
}

__global__ void insertMulKeys(KeyValue* hashtable, const KeyValue* kvs,
                              uint32_t numkvs) {
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < numkvs) {
    uint32_t key = kvs[tid].key;
    uint32_t value = kvs[tid].value;
    uint32_t slot = hash(key);
    while (true) {
      uint32_t prev = atomicCAS(&hashtable[slot].key, kEmpty, key);
      if (prev == kEmpty || prev == key) {
        hashtable[slot].value = value;
        printf("Inserted Key: %u, value: %u\n", key, value);
        return;
      }
      slot = (slot + 1) & (kHashTableCapacity - 1);
    }
  }
}

__device__ uint32_t lookupKey(KeyValue* hashtable, uint32_t key) {
  uint32_t slot = hash(key);
  while (true) {
    if (hashtable[slot].key == key) {
      return hashtable[slot].value;
    }
    if (hashtable[slot].key == kEmpty) {
      return kEmpty;
    }
    slot = (slot + 1) & (kHashTableCapacity + 1);
  }
}

__global__ void lookupMulKeys(KeyValue* hashtable, KeyValue* kvs,
                              unsigned int numkvs) {
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < numkvs) {
    uint32_t key = kvs[tid].key;
    uint32_t slot = hash(key);
    while (true) {
      if (hashtable[slot].key == key) {
        kvs[tid].value = hashtable[slot].value;
        return;
      }
      if (hashtable[slot].key == kEmpty) {
        kvs[tid].value = kEmpty;
        return;
      }
      slot = (slot + 1) & (kHashTableCapacity + 1);
    }
  }
}

KeyValue* create_hashtable() {
  KeyValue* hashtable;
  hipMalloc(&hashtable, sizeof(KeyValue) * kHashTableCapacity);
  checkCUDAError("HashTable initialization malloc");

  static_assert(kEmpty == 0xffffffff, "kEmpty has to be 0xffffffff");
  hipMemset(hashtable, 0xff, sizeof(KeyValue) * kHashTableCapacity);
  return hashtable;
}

std::vector<KeyValue> generate_random_KVs(std::mt19937& rnd, uint32_t numkvs) {
  std::uniform_int_distribution<uint32_t> dis(0, kEmpty - 1);
  std::vector<KeyValue> kvs;
  kvs.reserve(numkvs);

  for (uint32_t i = 0; i < numkvs; i++) {
    uint32_t key = dis(rnd);
    uint32_t value = dis(rnd);
    kvs.push_back(KeyValue{key, value});
  }
  return kvs;
}

/********************************/
std::vector<int>* readInput(const std::string& filename) {
  std::vector<int>* data = new std::vector<int>();
  std::ifstream file(filename);
  std::string line;
  int number;
  if (!file.is_open()) {
    std::cerr << "Error opening the file" << std::endl;
    return data;
  }

  while (getline(file, line)) {
    std::istringstream iss(line);
    while (iss >> number) {
      data->push_back(number);
    }
  }
  file.close();
  return data;
}
/********************************/
typedef struct Index {
  int* sorted_arr;
  KeyValue* map = nullptr;
} Index;

typedef struct Relation {
  char* name = nullptr;
  int num_rows;
  int num_cols;
  int num_indx_cols;
  int* index_col;
  Index index;
  int* data_arr;
} Relation;

void freeRelation_d(Relation* d_rel) {
  Relation* h_rel = (Relation*)malloc(sizeof(Relation));
  hipMemcpy(h_rel, d_rel, sizeof(Relation), hipMemcpyDeviceToHost);
  hipFree(h_rel->data_arr);
  hipFree(h_rel->index.sorted_arr);
  hipFree(h_rel->index.map);
  hipFree(h_rel->index_col);
  hipFree(h_rel->name);
  hipFree(d_rel);
  free(h_rel);
}
struct TupleLessCol {
  int col;
  const int* data_arr;

  TupleLessCol(int col, const int* data_arr) : col(col), data_arr(data_arr) {}

  __device__ bool operator()(const int& offset1, const int& offset2) const {
    return data_arr[offset1 + col] < data_arr[offset2 + col];
  }
};

struct TupleEqual {
  const int* data_arr;

  TupleEqual(const int* data_arr) : data_arr(data_arr) {}

  __device__ bool operator()(const int& x, const int& y) const {
    return (data_arr[x] == data_arr[y] && data_arr[x + 1] == data_arr[y + 1]);
  }
};

struct TupleLess {
  const int* data_arr1;
  const int* data_arr2;

  TupleLess(const int* data_arr1, const int* data_arr2)
      : data_arr1(data_arr1), data_arr2(data_arr2) {}

  __device__ bool operator()(const int& offset1, const int& offset2) {
    //    printf("1st ele:%d and 2nd ele: %d\n", data_arr1[offset1],
    //           data_arr2[offset2]);
    if (data_arr1[offset1] == data_arr2[offset2])
      return data_arr1[offset1 + 1] > data_arr2[offset2 + 1];
    else
      return data_arr1[offset1] > data_arr2[offset2];
  }
};

void printArray(int* arr, int count) {
  for (int i = 0; i < count; i++) {
    printf("%d\t", arr[i]);
  }
  printf("\n");
}

void printDeviceArray(int* d_arr, int size) {
  int* h_arr = (int*)malloc(sizeof(int) * size);
  hipMemcpy(h_arr, d_arr, sizeof(int) * size, hipMemcpyDeviceToHost);
  checkCUDAError("Inside printDeviceArray after cudaMempy\n");
  for (int i = 0; i < size; i++) {
    printf("%d\t", h_arr[i]);
  }
  printf("\n");
  free(h_arr);
}

int* copyDataArr(Relation* d_rel) {
  Relation* h_rel = (Relation*)malloc(sizeof(Relation));
  hipMemcpy(h_rel, d_rel, sizeof(Relation), hipMemcpyDeviceToHost);
  int* data_arr = (int*)malloc(sizeof(int) * h_rel->num_rows * 2);
  hipMemcpy(data_arr, h_rel->data_arr, sizeof(int) * h_rel->num_rows * 2,
             hipMemcpyDeviceToHost);
  return data_arr;
}

__global__ void d_printArray(int* arr, int size) {
  printf("printing array inside the device:\n");
  for (int i = 0; i < size; i++) {
    printf("%d\t", arr[i]);
  }
  printf("\n");
}
__global__ void initSortedArr(Relation* rel) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < rel->num_rows)
    rel->index.sorted_arr[idx] = idx * rel->num_cols;
}

__global__ void initMap(Relation* rel) {
  int cur_idx = threadIdx.x + blockDim.x * blockIdx.x;
  int prev_idx = cur_idx - 1;
  if (cur_idx < rel->num_rows) {
    if (cur_idx == 0 || (rel->data_arr[rel->index.sorted_arr[cur_idx]] !=
                         rel->data_arr[rel->index.sorted_arr[prev_idx]])) {
      uint32_t key = rel->data_arr[rel->index.sorted_arr[cur_idx]];
      uint32_t value = cur_idx;
      insertKey(rel->index.map, key, value);
    }
  }
}

__global__ void testKernel(Relation* rel) {
  printf("The Relation is %s\n", rel->name);
  printf("\t rel.numrows = %d\n", rel->num_rows);
  printf("\t rel.numcols= %d\n", rel->num_cols);
  printf("\t rel.num_indx_cols= %d\n", rel->num_indx_cols);
  for (int i = 0; i < rel->num_indx_cols; i++) {
    printf("%d\t", rel->index_col[i]);
  }
  printf("\n");
  printf("The data array is: \n");
  for (int i = 0; i < rel->num_rows; i++) {
    printf("%d\t%d", rel->data_arr[rel->index.sorted_arr[i]],
           rel->data_arr[rel->index.sorted_arr[i] + 1]);
    printf("\n");
  }
  printf("The sorted arrray is:\n");
  for (int i = 0; i < rel->num_rows; i++) {
    printf("%d ", rel->index.sorted_arr[i]);
  }
  printf("\n");
  //  printf("The map is:\n");
  //  for (int i = 0; i < kHashTableCapacity; i++) {
  //    printf("%d\t%d\n", rel->index.map[i].key, rel->index.map[i].value);
  //  }
}

__global__ void joinRelationCount(Relation* outer, Relation* inner,
                                  int* count_arr) {
  uint32_t idx = threadIdx.x + blockDim.x * blockIdx.x;
  int count = 0;
  if (idx < outer->num_rows) {
    int key = outer->data_arr[outer->index.sorted_arr[idx]];
    int inner_idx = lookupKey(inner->index.map, key);
    if (inner_idx != kEmpty) {
      for (int inner_srtd_indx = inner_idx; inner_srtd_indx < inner->num_rows;
           inner_srtd_indx++) {
        int inner_key =
            inner->data_arr[inner->index.sorted_arr[inner_srtd_indx]];
        if (inner_key == key)
          count++;
        else
          break;
      }
    }
  }
  if (idx < outer->num_rows)
    count_arr[idx] = count;
}

__global__ void joinRelationData(Relation* outer, Relation* inner,
                                 int* result_idx_arr, int* join_data_arr) {
  uint32_t idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < outer->num_rows) {
    uint32_t result_idx = 2 * result_idx_arr[idx];
    int key = outer->data_arr[outer->index.sorted_arr[idx]];
    int outer_value = outer->data_arr[outer->index.sorted_arr[idx] + 1];
    int inner_idx = lookupKey(inner->index.map, key);
    if (inner_idx != kEmpty) {
      for (int inner_srtd_indx = inner_idx; inner_srtd_indx < inner->num_rows;
           inner_srtd_indx++) {
        int inner_key =
            inner->data_arr[inner->index.sorted_arr[inner_srtd_indx]];
        int inner_value =
            inner->data_arr[inner->index.sorted_arr[inner_srtd_indx] + 1];
        if (inner_key == key) {
          //          if (DEBUG && idx == 2) {
          //            printf("The inner_key: %d, key:%d, result_idx:%d\n",
          //            inner_key, key,
          //                   result_idx);
          //          }
          join_data_arr[result_idx] = inner_value;
          join_data_arr[result_idx + 1] = outer_value;
          result_idx += 2;
        } else
          break;
      }
    }
  }
}

__global__ void makeDeltaData(Relation* new_rel, int* del_sorted_arr,
                              int del_num_rows, int* del_data_arr) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < del_num_rows) {
    int data_idx = del_sorted_arr[idx];
    del_data_arr[idx * 2] = new_rel->data_arr[data_idx];
    del_data_arr[idx * 2 + 1] = new_rel->data_arr[data_idx + 1];
  }
}

Relation* make_Relation(std::vector<int>* host_data_vec, int* d_data_inp,
                        int num_rows, int num_cols,
                        std::vector<int>* index_cols, char* name) {
  Relation* d_rel;
  Relation h_rel;
  hipMalloc((void**)&d_rel, sizeof(Relation));

  char* d_name;
  hipMalloc((void**)&d_name, strlen(name) + 1);
  hipMemcpy(d_name, name, strlen(name) + 1, hipMemcpyHostToDevice);

  int* d_data_arr;
  if (d_data_inp == nullptr) {
    hipMalloc((void**)&d_data_arr, sizeof(int) * host_data_vec->size());
    hipMemcpy(d_data_arr, host_data_vec->data(),
               host_data_vec->size() * sizeof(int), hipMemcpyHostToDevice);
  } else
    d_data_arr = d_data_inp;

  int* d_index_cols;
  hipMalloc((void**)&d_index_cols, sizeof(int) * index_cols->size());
  hipMemcpy(d_index_cols, index_cols->data(), index_cols->size() * sizeof(int),
             hipMemcpyHostToDevice);

  Index* d_index;
  hipMalloc((void**)&d_index, sizeof(Index));

  int* d_sorted_array;
  hipMalloc((void**)&d_sorted_array, sizeof(int) * num_rows);

  KeyValue* d_map = create_hashtable();

  h_rel.name = d_name;
  h_rel.num_rows = num_rows;
  h_rel.num_cols = num_cols;
  h_rel.num_indx_cols = index_cols->size();
  h_rel.index_col = d_index_cols;
  h_rel.index.map = d_map;
  h_rel.index.sorted_arr = d_sorted_array;
  h_rel.data_arr = d_data_arr;
  hipMemcpy(d_rel, &h_rel, sizeof(Relation), hipMemcpyHostToDevice);

  // make sorted array
  int blockSize = 256;
  int numBlocks = (num_rows + blockSize - 1) / blockSize;
  initSortedArr<<<numBlocks, blockSize>>>(d_rel);
  hipDeviceSynchronize();
  // sort sorted array
  thrust::device_ptr<int> t_data_arr(d_data_arr);
  thrust::device_ptr<int> t_sorted_arr(d_sorted_array);

  for (int col = num_cols - 1; col >= 0; col--) {
    TupleLessCol comp(col, d_data_arr);
    thrust::stable_sort(thrust::device, t_sorted_arr, t_sorted_arr + num_rows,
                        comp);
  }

  // make the map
  initMap<<<numBlocks, blockSize>>>(d_rel);
  hipDeviceSynchronize();
  return d_rel;
}

void removeDuplicates(Relation* rel) {
  Relation* h_rel = (Relation*)malloc(sizeof(Relation));
  hipMemcpy(h_rel, rel, sizeof(Relation), hipMemcpyDeviceToHost);

  thrust::device_ptr<int> t_new_srtd_arr(h_rel->index.sorted_arr);
  TupleEqual comp(h_rel->data_arr);
  auto new_end = thrust::unique(thrust::device, t_new_srtd_arr,
                                t_new_srtd_arr + h_rel->num_rows, comp);

  int new_size = new_end - t_new_srtd_arr;
  hipMemcpy(&(rel->num_rows), &new_size, sizeof(int), hipMemcpyHostToDevice);
}

Relation* joinRelations_host(Relation* outer, Relation* inner, int outer_rows) {
  int* h_count_arr = (int*)malloc(sizeof(int) * outer_rows);
  int* d_count_arr;
  hipMalloc((void**)&d_count_arr, sizeof(int) * outer_rows);

  int blockSize = 256;
  int numBlocks = (outer_rows + blockSize - 1) / blockSize;
  joinRelationCount<<<numBlocks, blockSize>>>(outer, inner, d_count_arr);
  hipDeviceSynchronize();

  thrust::device_ptr<int> t_count_arr(d_count_arr);
  thrust::exclusive_scan(t_count_arr, t_count_arr + outer_rows, t_count_arr);
  int totalJoinRowCount;
  hipMemcpy(&totalJoinRowCount, d_count_arr + outer_rows - 1, sizeof(int),
             hipMemcpyDeviceToHost);
  checkCUDAError("CudaMecpy after exclusive scan");
  if (totalJoinRowCount <= 0) {
    return nullptr;
  }
  if (DEBUG) {
    printf("Total Join Row count: %d\n", totalJoinRowCount);
  }
  //  printDeviceArray(d_count_arr, outer_rows);
  int* d_join_data_arr;
  hipMalloc((void**)&d_join_data_arr, sizeof(int) * 2 * totalJoinRowCount);
  joinRelationData<<<numBlocks, blockSize>>>(outer, inner, d_count_arr,
                                             d_join_data_arr);
  hipDeviceSynchronize();

  std::vector<int> index_cols{0};
  char path_new_name[] = "path_new";
  Relation* path_new =
      make_Relation(nullptr, d_join_data_arr, totalJoinRowCount, 2, &index_cols,
                    path_new_name);
  removeDuplicates(path_new);
  return path_new;
}

Relation* makeDelta(Relation* full_rel, Relation* new_rel) {
  if (DEBUG)
    printf("Inside the makeDelta\n");
  Relation* h_new_rel = (Relation*)malloc(sizeof(Relation));
  hipMemcpy(h_new_rel, new_rel, sizeof(Relation), hipMemcpyDeviceToHost);
  Relation* h_full_rel = (Relation*)malloc(sizeof(Relation));
  hipMemcpy(h_full_rel, full_rel, sizeof(Relation), hipMemcpyDeviceToHost);

  //  int* d_del_srtd_arr;
  //  hipMalloc((void**)&d_del_srtd_arr, sizeof(int) * h_new_rel->num_rows);
  //  checkCUDAError("hipMalloc for del_srtc_arr in makeDelta");
  //  thrust::device_ptr<int> t_new_rel_sorted(h_new_rel->index.sorted_arr);
  //  thrust::device_ptr<int> t_full_rel_sorted(h_full_rel->index.sorted_arr);
  //  thrust::device_ptr<int> t_del_srtd_arr(d_del_srtd_arr);
  //  if (DEBUG) {
  //    printf("Before set difference\n");
  //    printf("the number of rows new_rel:%d\tfull_rel:%d\n",
  //    h_new_rel->num_rows,
  //           h_full_rel->num_rows);
  //  }
  //
  //  TupleLess comp(h_new_rel->data_arr, h_full_rel->data_arr);
  //  auto del_end = thrust::set_difference(
  //      thrust::device, t_new_rel_sorted, t_new_rel_sorted +
  //      h_new_rel->num_rows, t_full_rel_sorted, t_full_rel_sorted +
  //      h_full_rel->num_rows, t_del_srtd_arr, comp);
  //  if (DEBUG)
  //    printf("After set difference\n");
  //  int delta_size = del_end - t_del_srtd_arr;
  //  if(DEBUG){
  //    printf("The delta size is:%d\n", delta_size);
  //    printf("the del_end:%d annd t_del_srtd_arr:%d\n", del_end,
  //    t_del_srtd_arr);
  //  }
  int delta_size = h_new_rel->num_rows;
  int* d_del_srtd_arr = h_new_rel->index.sorted_arr;
  int* d_del_data_arr;
  hipMalloc((void**)&d_del_data_arr, sizeof(int) * 2 * delta_size);
  checkCUDAError("hipMalloc for del_data_arrr in makeDelta");
  int blockSize = 256;
  int numBlocks = (delta_size + blockSize - 1) / blockSize;
  makeDeltaData<<<numBlocks, blockSize>>>(new_rel, d_del_srtd_arr, delta_size,
                                          d_del_data_arr);
  hipDeviceSynchronize();
  if (DEBUG)
    printf("After make Delta Data\n");
  checkCUDAError("Make Delta Data");
  std::vector<int> index_cols{0};
  char del_name[] = "path_delta";
  Relation* path_delta =
      make_Relation(nullptr, d_del_data_arr, delta_size, h_new_rel->num_cols,
                    &index_cols, del_name);

  if (DEBUG)
    printf("After making the Delta Relation in makeDelta\n");
  return path_delta;
}

Relation* updateFull(Relation* full_rel, Relation* del_rel) {
  //  printf("Inside update full");
  Relation* h_del_rel = (Relation*)malloc(sizeof(Relation));
  hipMemcpy(h_del_rel, del_rel, sizeof(Relation), hipMemcpyDeviceToHost);
  checkCUDAError("After the del_rel copy into h_del_rel\n");
  Relation* h_full_rel = (Relation*)malloc(sizeof(Relation));
  hipMemcpy(h_full_rel, full_rel, sizeof(Relation), hipMemcpyDeviceToHost);
  checkCUDAError("After the full_rel copy into h_full_rel\n");
  // I append the data arrays together, that should'nt be a problem.
  // I merge the sorted arrays
  // And do a linear scan and create the map.
  int* d_full_rel_merge;
  hipMalloc((void**)&d_full_rel_merge,
             sizeof(int) * 2 * (h_full_rel->num_rows + h_del_rel->num_rows));
  hipMemcpy(d_full_rel_merge, h_full_rel->data_arr,
             sizeof(int) * 2 * h_full_rel->num_rows, hipMemcpyDeviceToDevice);
  hipMemcpy(d_full_rel_merge + 2 * h_full_rel->num_rows, h_del_rel->data_arr,
             sizeof(int) * 2 * h_del_rel->num_rows, hipMemcpyDeviceToDevice);
  // this is a very inefficient, because we are basically discarding full_rel
  // sorted array and map
  std::vector<int> index_col{0};
  char path_full_name[] = "path_full";
  Relation* path_full = make_Relation(
      nullptr, d_full_rel_merge, (h_full_rel->num_rows + h_del_rel->num_rows),
      2, &index_col, path_full_name);
  return path_full;
}

int getRowCount(Relation* d_rel) {
  Relation* h_rel = (Relation*)malloc(sizeof(Relation));
  hipMemcpy(h_rel, d_rel, sizeof(Relation), hipMemcpyDeviceToHost);
  return h_rel->num_rows;
}

int main(int argc, char* argv[]) {
  std::vector<int> index_cols{0};
  std::string filename =
      "/data/user/home/skunapan/gpu_joins/gdlog/data/data_3.txt";
  if (argc > 1) {
    filename = std::string(argv[1]);
  }
  std::vector<int>* graph_edge = readInput(filename);
  if (DEBUG) {
    for (auto val : *graph_edge) {
      printf("%d\t", val);
    }
    printf("\n");
  }
  std::vector<int>* graph_path = readInput(filename);
  std::reverse(graph_path->begin(), graph_path->end());

  int num_cols = 2;
  int num_rows = graph_edge->size() / 2;
  printf("Rows: %d, Columns:%d\n", num_rows, num_cols);
  printf("\n");

  char edge_name[] = "edge";
  Relation* d_edge = make_Relation(graph_edge, nullptr, num_rows, num_cols,
                                   &index_cols, edge_name);
  if (DEBUG) {
    testKernel<<<1, 1>>>(d_edge);
    hipDeviceSynchronize();
  }
  char rel_name[] = "path";
  Relation* d_path = make_Relation(graph_path, nullptr, num_rows, num_cols,
                                   &index_cols, rel_name);
  if (DEBUG) {
    testKernel<<<1, 1>>>(d_path);
    hipDeviceSynchronize();
  }
  Relation *d_path_new, *d_path_delta, *d_path_full;
  d_path_delta = d_path;
  d_path_full = d_path;
  int count = -1;
  int iter_to_debug = -1;
  do {
    count++;
    if (DEBUG && count > iter_to_debug) {
      printf("---------------- %d Iteration --------------------\n", count);
      printf("thou shall debug this\n");
    }

    d_path_new =
        joinRelations_host(d_path_delta, d_edge, getRowCount(d_path_delta));
    if (d_path_new == nullptr) {
      break;
    }
//    freeRelation_d(d_path_delta);
    if (DEBUG && count > iter_to_debug) {
      printf("Joined the relations\n");
      testKernel<<<1, 1>>>(d_path_new);
      hipDeviceSynchronize();
    }
    d_path_delta = makeDelta(d_path_full, d_path_new);
    freeRelation_d(d_path_new);
    if (DEBUG && count > iter_to_debug) {
      printf("After make Delta\n");
      testKernel<<<1, 1>>>(d_path_delta);
      hipDeviceSynchronize();
    }
    Relation* d_path_full_temp = d_path_full;
    d_path_full = updateFull(d_path_full, d_path_delta);
    if (DEBUG && count > iter_to_debug) {
      printf("After updateFull\n");
      testKernel<<<1, 1>>>(d_path_full);
      hipDeviceSynchronize();
    }
    freeRelation_d(d_path_full_temp);
  } while (getRowCount(d_path_delta) != 0);

  printf("The number of Rows:%d\n", getRowCount(d_path_full));
  if (DEBUG) {
    int* final_data = copyDataArr(d_path_full);
    for (int i = 0; i < getRowCount(d_path_full); i++) {
      printf("%d\t%d\n", final_data[i * 2 + 1], final_data[i * 2]);
    }
    printf("\n");
  }
  freeRelation_d(d_path_delta);
  freeRelation_d(d_edge);
  freeRelation_d(d_path_full);
}
